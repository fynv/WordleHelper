#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include <algorithm>

#define WORD_LENGTH 5
#define NUM_WORDS 2309
#define MASK_LENGTH ((NUM_WORDS+31)>>5)

__constant__ int c_num_words;
__constant__ char c_words[NUM_WORDS][WORD_LENGTH];

inline __device__ void d_judge(int idx_truth, int idx_guess, char feedback[5])
{
	const char* truth = c_words[idx_truth];
	const char* guess = c_words[idx_guess];

	unsigned char mask_used = 0;

	for (int i = 0; i < 5; i++)
	{
		if (guess[i] == truth[i])
		{
			feedback[i] = 2;
			mask_used |= (1 << i);
		}
		else
		{
			feedback[i] = 0;
		}
	}

	for (int i = 0; i < 5; i++)
	{
		if (feedback[i] == 0)
		{
			for (int j = 0; j < 5; j++)
			{
				if ((mask_used & (1<<j)) == 0 && guess[i] == truth[j])
				{
					feedback[i] = 1;
					mask_used |= (1 << j);
					break;
				}
			}
		}
	}
}

inline __device__ int d_guess(int idx_truth, int idx_start)
{
	int masks_options[MASK_LENGTH];
	for (int i = 0; i < MASK_LENGTH; i++)
	{
		masks_options[i] = 0xFFFFFFFF;
	}

	unsigned masks_exclude[WORD_LENGTH];
	for (int i = 0; i < WORD_LENGTH; i++)
	{
		masks_exclude[i] = 0;		
	}

	int rounds = 0;
	while (true)
	{
		int idx_guess = idx_start;
		if (rounds > 0)
		{
			for (int i = 0; i < MASK_LENGTH; i++)
			{
				int ffs = __ffs(masks_options[i]);
				if (ffs > 0)
				{
					idx_guess = i * 32 + ffs - 1;
					break;
				}
			}
		}
		rounds++;

		if (idx_guess == idx_truth) break;

		char feedback[5];
		d_judge(idx_truth, idx_guess, feedback);

		unsigned char min_counts[26];
		unsigned char max_counts[26];
		for (int i = 0; i < 26; i++)
		{
			min_counts[i] = 0;
			max_counts[i] = 5;
		}

		const char* guess = c_words[idx_guess];

		for (int i = 0; i < 5; i++)
		{
			char c = guess[i];
			int j = feedback[i];

			if (j == 1)
			{
				min_counts[c - 'a']++;
				masks_exclude[i] |= (1 << (c - 'a'));
			}
			else if (j == 2)
			{
				min_counts[c - 'a']++;
				int count_set = __popc(masks_exclude[i]);
				if (count_set < 25)
				{
					for (int k = 0; k < 26; k++)
					{
						char c2 = 'a' + k;
						if (c2 != c)
						{
							masks_exclude[i] |= (1 <<(c2 -'a'));
						}
					}
				}
			}
			else
			{
				masks_exclude[i] |= (1 << (c - 'a'));
			}
		}


		for (int i = 0; i < 5; i++)
		{
			char c = guess[i];
			int j = feedback[i];
			if (j != 1 && j != 2)
			{
				max_counts[c - 'a'] = min_counts[c - 'a'];
			}
		}

		for (int i = 0; i < NUM_WORDS; i++)
		{
			int idx_mask = (i >> 5);
			int idx_bit = (i & 31);
			if ((masks_options[idx_mask] & (1 << idx_bit)) == 0) continue;

			const char* word = c_words[i];
			bool remove = false;

			unsigned char counts[26];
			for (int j = 0; j < 26; j++)
			{
				counts[j] = 0;
			}

			for (int j = 0; j < 5; j++)
			{
				char c = word[j];
				if ((masks_exclude[j] & (1 << (c - 'a'))) !=0)
				{
					remove = true;
					break;
				}				
				counts[c - 'a']++;
			}

			if (!remove)
			{
				for (int k = 0; k < 26; k++)
				{
					unsigned char min_count = min_counts[k];
					unsigned char max_count = max_counts[k];
					unsigned char count = counts[k];
					if (count<min_count || count>max_count)
					{
						remove = true;
						break;
					}
				}
			}

			if (remove)
			{
				masks_options[idx_mask] &= (~(1 << idx_bit));
			}

		}

	}
	return rounds;

}

__global__ void g_guess(int num_words, const int* idx_truths, int* rounds, int idx_start)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < num_words)
	{
		rounds[i] = d_guess(idx_truths[i], idx_start);
	}
}

__global__ void g_guess_all(int* rounds, int idx_start)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < c_num_words)
	{
		rounds[i] = d_guess(i, idx_start);
	}
}

__global__ void g_guess_matrix(int* rounds)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < c_num_words && j < c_num_words)
	{
		rounds[i + j* c_num_words] = d_guess(i, j);
	}
}


void h_set_words(const std::vector<std::string>& all_words)
{
	int num_words = (int)all_words.size();
	hipMemcpyToSymbol(HIP_SYMBOL(c_num_words), &num_words, sizeof(int));

	unsigned char h_words[NUM_WORDS][WORD_LENGTH];
	for (int i = 0; i < num_words; i++)
	{
		memcpy(h_words[i], all_words[i].c_str(), WORD_LENGTH);
	}
	hipMemcpyToSymbol(HIP_SYMBOL(c_words), h_words, num_words * WORD_LENGTH);

}

void h_guess(const std::vector<std::string>& all_words, int num_words, const std::string* words, int* rounds, const std::string& start_word)
{
	std::vector<int> idxs(num_words);
	for (int i = 0; i < num_words; i++)
	{
		idxs[i] = (int)(std::find(all_words.begin(), all_words.end(), words[i]) - all_words.begin());
	}

	int idx_start = (int)(std::find(all_words.begin(), all_words.end(), start_word) - all_words.begin());

	int* d_idxs;
	hipMalloc(&d_idxs, sizeof(int) * num_words);
	hipMemcpy(d_idxs, idxs.data(), sizeof(int) * num_words, hipMemcpyHostToDevice);

	int* d_rounds;
	hipMalloc(&d_rounds, sizeof(int) * num_words);
	
	int num_blocks = (num_words + 63) / 64;
	g_guess << < num_blocks, 64 >> > (num_words, d_idxs, d_rounds, idx_start);

	hipMemcpy(rounds, d_rounds, sizeof(int) * num_words, hipMemcpyDeviceToHost);

}

void h_guess_all(const std::vector<std::string>& all_words, int* rounds, const std::string& start_word)
{
	int num_words = (int)all_words.size();
	int idx_start = (int)(std::find(all_words.begin(), all_words.end(), start_word) - all_words.begin());

	int* d_rounds;
	hipMalloc(&d_rounds, sizeof(int) * num_words);

	int num_blocks = (num_words + 63) / 64;
	g_guess_all << < num_blocks, 64 >> > (d_rounds, idx_start);

	hipMemcpy(rounds, d_rounds, sizeof(int) * num_words, hipMemcpyDeviceToHost);

}

void h_guess_matrix(int num_words, int* rounds)
{
	int* d_rounds;
	hipMalloc(&d_rounds, sizeof(int) * num_words * num_words);

	unsigned num_blocks = (num_words + 7) / 8;
	dim3 blocks = { num_blocks , num_blocks ,1 };
	dim3 block_size = { 8 , 8 ,1 };
	g_guess_matrix << < blocks, block_size >> > (d_rounds);

	hipMemcpy(rounds, d_rounds, sizeof(int) * num_words * num_words, hipMemcpyDeviceToHost);

}